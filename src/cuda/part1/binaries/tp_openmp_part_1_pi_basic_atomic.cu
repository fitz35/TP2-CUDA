/*

This program will numerically compute the integral of

                  4/(1+x*x) 
				  
from 0 to 1.  The value of this integral is pi -- which 
is great since it gives us an easy way to check the answer.

History: Written by Tim Mattson, 11/1999.
         Modified/extended by Jonathan Rouzaud-Cornabas, 10/2022
*/


#include <hip/hip_runtime.h>
#include <limits>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <sys/time.h>
#include <fstream>

#define CHECK_CUDA_ERROR(x) checkCudaError(x, __FILE__, __LINE__)

static long num_steps = 100000000;
static int num_cores=1;
double step;

__global__ void pi_kernel(float *sum, int num_steps, double step, int steps_per_thread)
{
    int i;
    float x;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    for (i = id* steps_per_thread; i < id * steps_per_thread+steps_per_thread && i<num_steps ; i +=1) {
        x = (i + 0.5) * step;
        atomicAdd(sum, 4.0 / (1.0 + x * x));
    }
}

inline void checkCudaError(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        printf("CUDA error: %s:%d: %s)", file, line, hipGetErrorString(err));
        exit(-1);
    }
}

int main (int argc, char** argv)
{
    
      // Read command line arguments.
      for ( int i = 0; i < argc; i++ ) {
        if ( ( strcmp( argv[ i ], "-N" ) == 0 ) || ( strcmp( argv[ i ], "-num_steps" ) == 0 ) ) {
            num_steps = atol( argv[ ++i ] );
            printf( "  User num_steps is %ld\n", num_steps );
        }
        if ( ( strcmp( argv[ i ], "-C" ) == 0 ) || ( strcmp( argv[ i ], "-num_cores" ) == 0 ) ) {
            num_cores = atol( argv[ ++i ] );
            printf( "  User num_cores is %d\n", num_cores );
        
        } else if ( ( strcmp( argv[ i ], "-h" ) == 0 ) || ( strcmp( argv[ i ], "-help" ) == 0 ) ) {
            printf( "  Pi Options:\n" );
            printf( "  -num_steps (-N) <int>:      Number of steps to compute Pi (by default 100000000)\n" );
            printf( "  -help (-h):            print this message\n\n" );
            exit( 1 );
        }
      }
	  double  pi;
      float *sum= (float * )malloc(sizeof(float));
      sum[0]= 0; 
      step = 1.0/(double) num_steps;

      // Timer products.
      struct timeval begin, end;
      
      //Allocate memory on the device
        float *d_sum;      
        CHECK_CUDA_ERROR(hipMalloc((void **)&d_sum, sizeof(float)));
        CHECK_CUDA_ERROR(hipMemcpy(d_sum, sum, sizeof(float), hipMemcpyHostToDevice));

      gettimeofday( &begin, NULL );
        // Launch kernel
        pi_kernel<<<num_cores, 1>>>(d_sum, num_steps, step, floor(num_steps/num_cores) );
        hipDeviceSynchronize();
        gettimeofday( &end, NULL );

        // Copy result back to host
        CHECK_CUDA_ERROR(hipMemcpy(sum, d_sum, sizeof(float), hipMemcpyDeviceToHost));
        
	 

	  pi = step * sum[0];

      
     

      // Calculate time.
      double time = 1.0 * ( end.tv_sec - begin.tv_sec ) +
                1.0e-6 * ( end.tv_usec - begin.tv_usec );

    free(sum);
    hipFree(d_sum);
                
      printf("\n pi with %ld steps is %lf in %lf seconds\n ",num_steps,pi,time);
      std::ofstream myfile;
      myfile.open ("../pi_Stats.csv", std::ios_base::app);
      myfile << "Basic_Atomic,"<< num_steps << "," << num_cores << "," << time << "," << pi<<std::endl ;
      myfile.close();
      return 0;
}
